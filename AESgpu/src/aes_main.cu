#include "hip/hip_runtime.h"
#include <aes_core.h>
#include <aes_tables.h>
#include <common.h>

#include <cassert>

__constant__ AES_KEY globalKey;

__global__ void AES_ecb_encrypt_kernel(
        const u8     *in_d,
        u8           *out_d,
        unsigned long block_count,
        u64          *timeTaken_d
) {
    /* computer the thread id */
    int        idx = blockDim.x * blockIdx.x + threadIdx.x;
    const u32 *rk;
    u32        s0, s1, s2, s3, t0, t1, t2, t3;
    const u8  *in  = in_d + 16 * idx;
    u8        *out = out_d + 16 * idx;
    assert(in && out);
    rk = globalKey.rd_key;
    __syncthreads();
    u64 beginning = clock64();
    /*
     * map byte array block to cipher state
     * and add initial round key:
     */
    s0 = GETU32(in) ^ rk[0];
    s1 = GETU32(in + 4) ^ rk[1];
    s2 = GETU32(in + 8) ^ rk[2];
    s3 = GETU32(in + 12) ^ rk[3];
    /* round 1: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >> 8) & 0xff] ^
         Te3[s3 & 0xff] ^ rk[4];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >> 8) & 0xff] ^
         Te3[s0 & 0xff] ^ rk[5];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >> 8) & 0xff] ^
         Te3[s1 & 0xff] ^ rk[6];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >> 8) & 0xff] ^
         Te3[s2 & 0xff] ^ rk[7];
    /* round 2: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >> 8) & 0xff] ^
         Te3[t3 & 0xff] ^ rk[8];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >> 8) & 0xff] ^
         Te3[t0 & 0xff] ^ rk[9];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >> 8) & 0xff] ^
         Te3[t1 & 0xff] ^ rk[10];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >> 8) & 0xff] ^
         Te3[t2 & 0xff] ^ rk[11];
    /* round 3: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >> 8) & 0xff] ^
         Te3[s3 & 0xff] ^ rk[12];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >> 8) & 0xff] ^
         Te3[s0 & 0xff] ^ rk[13];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >> 8) & 0xff] ^
         Te3[s1 & 0xff] ^ rk[14];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >> 8) & 0xff] ^
         Te3[s2 & 0xff] ^ rk[15];
    /* round 4: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >> 8) & 0xff] ^
         Te3[t3 & 0xff] ^ rk[16];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >> 8) & 0xff] ^
         Te3[t0 & 0xff] ^ rk[17];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >> 8) & 0xff] ^
         Te3[t1 & 0xff] ^ rk[18];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >> 8) & 0xff] ^
         Te3[t2 & 0xff] ^ rk[19];
    /* round 5: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >> 8) & 0xff] ^
         Te3[s3 & 0xff] ^ rk[20];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >> 8) & 0xff] ^
         Te3[s0 & 0xff] ^ rk[21];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >> 8) & 0xff] ^
         Te3[s1 & 0xff] ^ rk[22];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >> 8) & 0xff] ^
         Te3[s2 & 0xff] ^ rk[23];
    /* round 6: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >> 8) & 0xff] ^
         Te3[t3 & 0xff] ^ rk[24];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >> 8) & 0xff] ^
         Te3[t0 & 0xff] ^ rk[25];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >> 8) & 0xff] ^
         Te3[t1 & 0xff] ^ rk[26];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >> 8) & 0xff] ^
         Te3[t2 & 0xff] ^ rk[27];
    /* round 7: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >> 8) & 0xff] ^
         Te3[s3 & 0xff] ^ rk[28];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >> 8) & 0xff] ^
         Te3[s0 & 0xff] ^ rk[29];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >> 8) & 0xff] ^
         Te3[s1 & 0xff] ^ rk[30];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >> 8) & 0xff] ^
         Te3[s2 & 0xff] ^ rk[31];
    /* round 8: */
    s0 = Te0[t0 >> 24] ^ Te1[(t1 >> 16) & 0xff] ^ Te2[(t2 >> 8) & 0xff] ^
         Te3[t3 & 0xff] ^ rk[32];
    s1 = Te0[t1 >> 24] ^ Te1[(t2 >> 16) & 0xff] ^ Te2[(t3 >> 8) & 0xff] ^
         Te3[t0 & 0xff] ^ rk[33];
    s2 = Te0[t2 >> 24] ^ Te1[(t3 >> 16) & 0xff] ^ Te2[(t0 >> 8) & 0xff] ^
         Te3[t1 & 0xff] ^ rk[34];
    s3 = Te0[t3 >> 24] ^ Te1[(t0 >> 16) & 0xff] ^ Te2[(t1 >> 8) & 0xff] ^
         Te3[t2 & 0xff] ^ rk[35];
    /* round 9: */
    t0 = Te0[s0 >> 24] ^ Te1[(s1 >> 16) & 0xff] ^ Te2[(s2 >> 8) & 0xff] ^
         Te3[s3 & 0xff] ^ rk[36];
    t1 = Te0[s1 >> 24] ^ Te1[(s2 >> 16) & 0xff] ^ Te2[(s3 >> 8) & 0xff] ^
         Te3[s0 & 0xff] ^ rk[37];
    t2 = Te0[s2 >> 24] ^ Te1[(s3 >> 16) & 0xff] ^ Te2[(s0 >> 8) & 0xff] ^
         Te3[s1 & 0xff] ^ rk[38];
    t3 = Te0[s3 >> 24] ^ Te1[(s0 >> 16) & 0xff] ^ Te2[(s1 >> 8) & 0xff] ^
         Te3[s2 & 0xff] ^ rk[39];

    // move the pointer 16 * 10 bytes ahead to get to the final round key
    rk += globalKey.rounds << 2;
    /*
     * apply last round and
     * map cipher state to byte array block:
     */
    s0 = (Te4[(t0 >> 24)] & 0xff000000) ^ (Te4[(t1 >> 16) & 0xff] & 0x00ff0000) ^
         (Te4[(t2 >> 8) & 0xff] & 0x0000ff00) ^ (Te4[(t3) &0xff] & 0x000000ff) ^
         rk[0];
    PUTU32(out,
           s0);
    s1 = (Te4[(t1 >> 24)] & 0xff000000) ^ (Te4[(t2 >> 16) & 0xff] & 0x00ff0000) ^
         (Te4[(t3 >> 8) & 0xff] & 0x0000ff00) ^ (Te4[(t0) &0xff] & 0x000000ff) ^
         rk[1];
    PUTU32(out + 4,
           s1);
    s2 = (Te4[(t2 >> 24)] & 0xff000000) ^ (Te4[(t3 >> 16) & 0xff] & 0x00ff0000) ^
         (Te4[(t0 >> 8) & 0xff] & 0x0000ff00) ^ (Te4[(t1) &0xff] & 0x000000ff) ^
         rk[2];
    PUTU32(out + 8,
           s2);
    s3 = (Te4[(t3 >> 24)] & 0xff000000) ^ (Te4[(t0 >> 16) & 0xff] & 0x00ff0000) ^
         (Te4[(t1 >> 8) & 0xff] & 0x0000ff00) ^ (Te4[(t2) &0xff] & 0x000000ff) ^
         rk[3];
    PUTU32(out + 12,
           s3);
    u64 end      = clock64();
    *timeTaken_d = (end - beginning);
}

/**
 * AES ECB Encryption wrapper for the underlying CUDA kernel
 * Takes a 32 block (1 block = 128 bits) plaintext and encrypts it.
 * Each block is handled by a separate thread -> 32 threads for 32 blocks (1
 * warp)
 */
void AES_ecb_128_encrypt_gpu(
        const u8          *in_h,
        u8                *out_h,
        const AES_KEY     *expanded_key_h,
        u64               *timeTaken_h,
        hipStream_t       stream,
        unsigned long      aes_block_count,
        const unsigned int threads_per_blk
) {
    u8  *in_d;
    u8  *out_d;
    u64 *timeTaken_d;
    hipMalloc(
            &in_d,
            aes_block_count * 16
    );// 1 block -> 128 bits -> 16 bytes
    hipMalloc(
            &out_d,
            aes_block_count * 16
    );// 1 block -> 128 bits -> 16 bytes

    hipMalloc(
            &timeTaken_d,
            8
    );// allocate 4 bytes to store timing information

    // copy the input data to the GPU
    hipMemcpy(
            in_d,
            in_h,
            aes_block_count * 16,
            hipMemcpyHostToDevice
    );

    // copy the expanded key to the GPU's constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(
            globalKey),
            expanded_key_h,
            sizeof(AES_KEY)
    );

    unsigned int num_cuda_blks =
            (aes_block_count + threads_per_blk - 1) / threads_per_blk;// ceil
    if (stream == 0) {
        AES_ecb_encrypt_kernel<<<num_cuda_blks, threads_per_blk>>>(
                in_d,
                out_d,
                aes_block_count,
                timeTaken_d
        );
    } else {
        AES_ecb_encrypt_kernel<<<num_cuda_blks, threads_per_blk, 0, stream>>>(
                in_d,
                out_d,
                aes_block_count,
                timeTaken_d
        );
    }
    hipMemcpy(
            out_h,
            out_d,
            aes_block_count * 16,
            hipMemcpyDeviceToHost
    );
    hipMemcpy(
            timeTaken_h,
            timeTaken_d,
            8,
            hipMemcpyDeviceToHost
    );
    hipFree(in_d);
    hipFree(out_d);
    hipFree(timeTaken_d);
}

int AES_set_encrypt_key_128(
        const unsigned char *userKey,
        AES_KEY             *key
) {
    u32 *rk;
    int  i = 0;
    u32  temp;

    if (!userKey || !key)
        return -1;
    key->rounds = 10;
    rk          = key->rd_key;
    rk[0]       = GETU32(userKey);
    rk[1]       = GETU32(userKey + 4);
    rk[2]       = GETU32(userKey + 8);
    rk[3]       = GETU32(userKey + 12);
    for (;;) {
        temp  = rk[3];
        rk[4] = rk[0] ^ (Te4_host[(temp >> 16) & 0xff] & 0xff000000) ^
                (Te4_host[(temp >> 8) & 0xff] & 0x00ff0000) ^
                (Te4_host[(temp) &0xff] & 0x0000ff00) ^
                (Te4_host[(temp >> 24)] & 0x000000ff) ^ rcon[i];
        rk[5] = rk[1] ^ rk[4];
        rk[6] = rk[2] ^ rk[5];
        rk[7] = rk[3] ^ rk[6];
        if (++i == 10) {
            return 0;
        }
        rk += 4;
    }
    return 0;
}