
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <algorithm>
using namespace std;
int numCacheLines = 6; // num of blocks
int stride = 64;       // value for the cache line size
int const max_space = sizeof(int) * 32; // allocating space for 32 different indices, each index is of type int
__device__ long long int tempVar = 0;
__global__ void testKernel(char *A, int *indices, char *result, int *_time)
{
    int index = indices[threadIdx.x];
    *_time = clock();
    result[threadIdx.x] = A[index];
    *_time = clock() - *_time;
}

__global__ void clearKernel(int *B)
{
    int result = B[threadIdx.x];
    tempVar += result;
}

// generating all the indices
void setIndices(int *local_indices)
{
    int curCacheLineIndex = 0;
    for (int i = 1; i <= 32; i++)
    {
        local_indices[i - 1] = curCacheLineIndex * stride;
        if (curCacheLineIndex < numCacheLines - 1)
        {
            curCacheLineIndex++;
        }
    }
    random_shuffle(local_indices, local_indices + 32);
}

/**
 * Command-line args:
 * argv[1] -> stride (32, 64, 128)
 * argv[2] -> output file
 * argv[3] -> number of tests
 */
int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        cerr << "Need 2 arguments" << endl;
        return -1;
    }
    stride = stoi(argv[1]);
    ofstream outputTable(argv[2]);
    int num_tests = stoi(argv[3]);

    char *A;
    int *indices;
    char *result;
    int *_time;
    hipMalloc((char **)&A, 10 * 1024 * 1024); // Allocating 10 MiB
    hipMalloc((int **)&indices, max_space);  // 32 indices for 32 threads at most
    hipMalloc((char **)&result, max_space);   // same logic as above
    hipMalloc((int **)&_time, sizeof(int));
    hipMemset(indices, 0, max_space);
    int *local_indices = (int *)malloc(max_space);

    int *garbage;
    hipMalloc((int **)&garbage, 10 * 1024 * 1024); // Allocating 10MiB of garbage

    for (int j = 5; j <= 32; j++)
    {
        numCacheLines = j;
        for (int i = 0; i < num_tests; i++)
        {
            memset(local_indices, 0, max_space);
            setIndices(local_indices);
            hipMemcpy(indices, local_indices, max_space, hipMemcpyHostToDevice);
            clearKernel<<<1, 32>>>(garbage); // warm up GPU
            testKernel<<<1, 32>>>(A, indices, result, _time);
            hipDeviceSynchronize();
            int local_time;
            hipMemcpy(&local_time, _time, sizeof(int), hipMemcpyDeviceToHost);
            outputTable << j << " " << local_time << endl;
        }
    }

    hipFree(A);
    hipFree(result);
    hipFree(indices);
    hipFree(_time);
}